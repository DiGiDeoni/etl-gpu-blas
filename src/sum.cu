//=======================================================================
// Copyright (c) 2017 Baptiste Wicht
// Distributed under the terms of the MIT License.
// (See accompanying file LICENSE or copy at
//  http://opensource.org/licenses/MIT)
//=======================================================================

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "egblas/sum.hpp"
#include "egblas/assert.hpp"

float egblas_ssum(float* x, size_t n, size_t s){
    egblas_assert(s == 1, "Stride is not yet supported for egblas_ssum");
    egblas_unused(s);

    return thrust::reduce(thrust::device, x, x + n);
}

double egblas_dsum(double* x, size_t n, size_t s){
    egblas_assert(s == 1, "Stride is not yet supported for egblas_dsum");
    egblas_unused(s);

    return thrust::reduce(thrust::device, x, x + n);
}

struct single_complex_add {
    __device__ hipComplex operator()(hipComplex x, hipComplex y){
        return hipCaddf(x, y);
    }
};

struct double_complex_add {
    __device__ hipDoubleComplex operator()(hipDoubleComplex x, hipDoubleComplex y){
        return hipCadd(x, y);
    }
};

hipComplex egblas_csum(hipComplex* x, size_t n, size_t s){
    egblas_assert(s == 1, "Stride is not yet supported for egblas_csum");
    egblas_unused(s);

    return thrust::reduce(thrust::device, x, x + n, make_hipComplex(0, 0), single_complex_add());
}

hipDoubleComplex egblas_zsum(hipDoubleComplex* x, size_t n, size_t s){
    egblas_assert(s == 1, "Stride is not yet supported for egblas_zsum");
    egblas_unused(s);

    return thrust::reduce(thrust::device, x, x + n, make_hipDoubleComplex(0, 0), double_complex_add());
}
